
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the the GPU\n");
}


int main(void)
{
    hello_from_gpu<<<1, 4>>>();
    hipDeviceSynchronize();

    return 0;
}
