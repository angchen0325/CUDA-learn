
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;

    const int id = threadIdx.x + blockIdx.x * blockDim.x; 
    printf("Hello World from block %d and thread %d, global id %d of GPU\n", bid, tid, id);
}


int main(void)
{
    hello_from_gpu<<<13, 6>>>();
    hipDeviceSynchronize();

    return 0;
}