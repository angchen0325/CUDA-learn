
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("'Hello World' from the the GPU\n");
}


int main(void)
{
    hello_from_gpu<<<3, 3>>>();
    hipDeviceSynchronize();

    return 0;
}